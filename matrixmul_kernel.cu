#include "hip/hip_runtime.h"
#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixMul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for Matrix Multiplication functionality
//! @param M  input data
//! @param N  input data
//! @param P  output data
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix* M, Matrix* N, Matrix* P)
{
	// thread index
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	
	int *Mptr = (int*)((void*)M);
	int *Nptr = (int*)((void*)N);
	int *Pptr = (int*)((void*)P);
	
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	
	int row = blockDim.y * by + ty;
	int col = blockDim.x * bx + tx;
	
	float Pvalue = 0;
	
	for (int k = 0; k < WIDTH; k++)
        {
            Pvalue += Mptr[row*WIDTH+k] * Nptr[k*WIDTH+col];
        }
	Pptr[row*WIDTH+col] = Pvalue;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
