
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


// includes, project
#include <cutil.h>

// includes, kernels
#include <matrixmul_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void MatrixMulOnDevice(Matrix* M, Matrix* N, Matrix* P)
{
	//Interface host call to the device kernel code and invoke the kernel
	 Matrix *Md, *Nd, *Pd;
	 
	//Step 1 allocate memory using cuda malloc
	hipMalloc( (void**) &Md, sizeof(Matrix) );
	hipMalloc( (void**) &Nd, sizeof(Matrix) );
	hipMalloc( (void**) &Pd, sizeof(Matrix) );
	
	//Step 2 memcpy to GPU from host using cudamemcpy
	 hipMemcpy(Md, M, sizeof(Matrix), hipMemcpyHostToDevice);
	 hipMemcpy(Nd, N, sizeof(Matrix), hipMemcpyHostToDevice);
	 hipMemcpy(Pd, P, sizeof(Matrix), hipMemcpyHostToDevice);

	
	// step 3: setup the threads should be based on some constant	
	
	int width, height;
	
	//this seems wrong
	width = height = KHEIGHT;
	
	//http://www.cs.nyu.edu/manycores/cuda_many_cores.pdf see page 17
	//Dimensions of the block in threads
	dim3 bDim(width,height);
	
	//Dimensions of the grid in blocks
	dim3 gDim(width/bDim.x,height/bDim.y);
	
	//Step 4 call the kernel
	 MatrixMulKernel<<<gDim,bDim >>>(Md,Nd,Pd);
	
	//Step 5 memcopy computed value back from device to host
	 hipMemcpy(P, Pd, sizeof(Matrix), hipMemcpyDeviceToHost);
	 
	 //Step 6 free the malloced memory of step 1
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	
	
	
}

int main()
{
  return 0;
}